#include "hip/hip_runtime.h"
#include <assert.h>
#include <geometry/sphere.h>
#include <math.h>
#include <stdio.h>

__device__ bool ray_sphere_hit(struct Sphere sphere, struct Ray ray, float t_min, float t_max, HitRecord* out_rec)
{
    const vec3 oc = ray.origin - sphere.origin;
    float const a = vec3_length_squared(ray.direction);
    float const half_b = vec3_dot(oc, ray.direction);
    float const c = vec3_length_squared(oc) - sphere.radius * sphere.radius;

    float const discriminant = half_b * half_b - a * c;
    if (discriminant < 0)
        return false;
    float const sqrtd = sqrt(discriminant);

    // Find the nearest root that lies in the acceptable range
    float root = (-half_b - sqrtd) / a;
    if (root < t_min || t_max < root) {
        root = (-half_b + sqrtd) / a;
        if (root < t_min || t_max < root)
            return false;
    }

    out_rec->t = root;
    out_rec->p = ray_at(ray, out_rec->t);
    out_rec->material = sphere.material;
    const vec3 outward_normal = (out_rec->p - sphere.origin) / sphere.radius;
    set_face_normal(out_rec, ray, outward_normal);

    float const theta = acos(-outward_normal.y);
    float const phi = atan2(-outward_normal.z, outward_normal.x) + M_PI;
    out_rec->u = phi / (2 * M_PI);
    out_rec->v = theta / M_PI;

    return true;
}
