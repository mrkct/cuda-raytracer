#include <materials/lambertian.h>
#include <scenes/single_sphere.h>
#include <textures/checkered_texture.h>

struct Scene create_single_sphere_scene(void)
{
    struct Scene scene;

    Sphere* sphere;
    Material* material;
    LambertianData* material_data;
    Texture* texture;
    CheckeredTextureData* texture_data;

    hipMallocManaged(&material, sizeof(*material));
    hipMallocManaged(&material_data, sizeof(*material_data));
    hipMallocManaged(&sphere, sizeof(*sphere));
    hipMallocManaged(&texture, sizeof(*texture));
    hipMallocManaged(&texture_data, sizeof(*texture_data));

    *texture_data = make_checkered_texture_data(0.1, make_color(0.8, 0, 0), make_color(0.9, 0.9, 0.9));
    *texture = make_checkered_texture(texture_data);
    *material_data = make_lambertian_material_data(texture);
    *material = make_lambertian_material(material_data);
    *sphere = make_sphere(make_vec3(0, 0, 2), 1, material);

    scene.spheres = sphere;
    scene.spheres_length = 1;

    return scene;
}